#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include "common.h"
#include "bmp.h"
#include <stdlib.h>
#include <GL/freeglut.h>

#define DIM 512
#define blockSize 8

size_t size;

#define PI 3.1415926535897932f
#define centerX (DIM/2)
#define centerY (DIM/2)

float sourceColors[DIM*DIM];	// host memory for source image
float readBackPixels[DIM*DIM];	// host memory for swirled image

float *sourceDevPtr;			// device memory for source image
float *swirlDevPtr;				// device memory for swirled image

__global__ void swirlKernel( float *sourcePtr, float *targetPtr ) 
{
	int index = 0;
    // TODO: Index berechnen	
	//int X = threadIdx.x + blockIdx.x * blockDim.x;
	//int Y = threadIdx.y + blockIdx.y * blockDim.y;
	//index = X + Y * blockDim.x;

	index = threadIdx.x;

	// TODO: Den swirl invertieren.

	targetPtr[index] = sourcePtr[index];    // simple copy
}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// TODO: Swirl Kernel aufrufen.
	dim3 grid(blockSize, blockSize);
	dim3 block(blockSize*blockSize, blockSize*blockSize);
	//swirlKernel<<<grid,block>>>(sourceDevPtr, swirlDevPtr);
	swirlKernel<<<1,DIM*DIM>>>(sourceDevPtr, swirlDevPtr);

	// TODO: Ergebnis zu host memory zuruecklesen.
	CUDA_SAFE_CALL( hipMemcpy(readBackPixels, swirlDevPtr, size, hipMemcpyDeviceToHost) );

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels( DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels );

	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
    CUDA_SAFE_CALL( hipFree( sourceDevPtr ) ); 
    CUDA_SAFE_CALL( hipFree( swirlDevPtr ) ); 
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Simple OpenGL CUDA");
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// load bitmap	
	Bitmap bmp = Bitmap("who-is-that.bmp");
	if (bmp.isValid())
	{		
		for (int i = 0 ; i < DIM*DIM ; i++) {
			sourceColors[i] = bmp.getR(i/DIM, i%DIM) / 255.0f;
		}
	}

	// TODO: allocate memory at sourceDevPtr on the GPU and copy sourceColors into it.
	size = DIM * DIM * sizeof(float);

	CUDA_SAFE_CALL( hipMalloc((void**)&sourceDevPtr, size) );
	CUDA_SAFE_CALL( hipMemcpy(sourceDevPtr, sourceColors, size, hipMemcpyHostToDevice) );
	
	// TODO: allocate memory at swirlDevPtr for the unswirled image.	
	CUDA_SAFE_CALL( hipMalloc((void**)&swirlDevPtr, size) );

	glutMainLoop();

	cleanup();
}
