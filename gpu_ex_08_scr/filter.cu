#include "hip/hip_runtime.h"

#include "common.h"
#include <stdlib.h>
#include <GL/freeglut.h>

#define DIM 512
#define blockSize 8
#define blurRadius 6
#define effectiveBlockSize (blockSize+2*blurRadius)

float sourceColors[DIM*DIM];
float *sourceDevPtr;
float *transDevPtr;

float readBackPixels[DIM*DIM];

int timer = 0;

void keyboard(unsigned char key, int x, int y)
{
	
}

__global__ void animateKernel( float *sourcePtr, float *targetPtr, int time) 
{
	int index = 0;
    // TODO: Index berechnen	
	int X = threadIdx.x + blockIdx.x * blockDim.x;
	int Y = threadIdx.y + blockIdx.y * blockDim.y;
	index = X + Y * blockDim.x * gridDim.x;

	int transX = X;
	transX += time%DIM;
	if(transX >= DIM) transX -= DIM;

	int transIndex = transX + Y * blockDim.x * gridDim.x;

	targetPtr[index] = sourcePtr[transIndex];    // simple copy
}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// TODO: Transformationskernel auf sourceDevPtr anwenden
	dim3 grid(DIM/blockSize, DIM/blockSize);
	dim3 block(blockSize, blockSize);

	timer += 1;

	animateKernel<<<grid,block>>>(sourceDevPtr, transDevPtr, timer);

	// TODO: Zeitmessung starten (see hipEventCreate, hipEventRecord)

	// TODO: Kernel mit Blur-Filter ausf�hren.

	// TODO: Zeitmessung stoppen und fps ausgeben (see hipEventSynchronize, hipEventElapsedTime, hipEventDestroy)

	// Ergebnis zur CPU zuruecklesen
    CUDA_SAFE_CALL( hipMemcpy( readBackPixels,
                              transDevPtr,
                              DIM*DIM*4,
                              hipMemcpyDeviceToHost ) );

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels( DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels );
	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
    CUDA_SAFE_CALL( hipFree( sourceDevPtr ) );     
	// TODO: Aufr�umen zus�tzlich angelegter Ressourcen.
	CUDA_SAFE_CALL( hipFree( transDevPtr ) );  
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Memory Types");
	glutKeyboardFunc(keyboard);
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// mit Schachbrettmuster f�llen
	for (int i = 0 ; i < DIM*DIM ; i++) {

		int x = (i % DIM) / (DIM/8);
		int y = (i / DIM) / (DIM/8);

		if ((x + y) % 2)
			sourceColors[i] = 1.0f;
		else
			sourceColors[i] = 0.0f;
	}

	// alloc memory on the GPU
	CUDA_SAFE_CALL( hipMalloc( (void**)&sourceDevPtr, DIM*DIM*4 ) );
    CUDA_SAFE_CALL( hipMemcpy( sourceDevPtr, sourceColors, DIM*DIM*4, hipMemcpyHostToDevice ) );

	// TODO: Weiteren Speicher auf der GPU f�r das Bild nach der Transformation und nach dem Blur allokieren.
	CUDA_SAFE_CALL( hipMalloc( (void**)&transDevPtr, DIM*DIM*4 ) );

	// TODO: Binding des Speichers des Bildes an eine Textur mittels hipBindTexture.

	glutMainLoop();

	cleanup();
}
